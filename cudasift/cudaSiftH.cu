#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaSift.cuh"


void InitCuda(int devNum) {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if (nDevices) {
        std::cerr << "No CUDA devices available" << std::endl;
        return ;
    }

    devNum = std::min(nDevices - 1, devNum);
    deviceInit(devNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devNum);
    std::cout << "Device Number: " << devNum << std::endl;

    return ;
}

