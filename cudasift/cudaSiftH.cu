#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaSift.h"
#include "cudautils.h"
#include "cudaSiftD.h"
#include "cudaSiftH.h"

#include "cudaSiftD.cu"


void InitCuda(int devNum) {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if (!nDevices) {
        std::cerr << "No CUDA devices available" << std::endl;
        return ;
    }

    devNum = std::min(nDevices - 1, devNum);
    deviceInit(devNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devNum);
    std::cout << "\t Device Number: " << devNum << std::endl;
    std::cout << "\t Memory name: " << prop.name << std::endl;
    std::cout << "\t Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
    printf("\t Peak Memory Bandwidth (GB/s): %.1f\n\n",
	            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    return ;
}

float *AllocSiftTempMemory(int width, int height, int numOctaves, bool scaleUp) {
    TimerGPU timer(0);
    const int nd = NUM_SCALES + 3;
    int w = width * (scaleUp ? 2 : 1);
    int h = height * (scaleUp ? 2 : 1);
    int p = iAlignUp(w, 128);
    int size = h * p;           // image sizes
    int sizeTmp = nd * h * p;   // laplace buffer sizes
    for (int i = 0; i < numOctaves; i++) {
        w /= 2;
        h /= 2;
        int p = iAlignUp(w, 128);
        size += h * p;
        sizeTmp += nd * h * p;
    }
    float *memoryTmp = nullptr;
    size_t pitch;
    size += sizeTmp;
    safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size + 4095) / 4096 * sizeof(float)));
#ifndef VERBOSE
    printf("Allocated memory size: %d bytes\n", size);
    printf("Memory allocated time = \t %.2f ms \n\n", timer.read());
#endif
    return memoryTmp;
}

/* Multi-scale functions */
void PrepareLaplaceKernels(int numOctaves, float initBlur, float *kernel) {
    if (numOctaves > 1) {
        float totInitBlur = (float)sqrt(initBlur * initBlur + 0.5f * 0.5f) / 2.0f;
        PrepareLaplaceKernels(numOctaves - 1, totInitBlur, kernel);
    }
    float scale = pow(2.0f, -1.0f / NUM_SCALES);
    float diffScale = pow(2.0f, 1.0f / NUM_SCALES);
    for (int i = 0; i < NUM_SCALES + 3; i++) {
        float kernelSum = 0.0f;
        float var = scale * scale - initBlur * initBlur;
        for (int j = 0; j <= LAPLACE_R; j++) {
            kernel[numOctaves * 12 * 16 + 16 * i + j] = (float)expf(-(double)j *j / 2.0 / var);
            kernelSum += (j == 0 ? 1 : 2) * kernel[numOctaves * 12 * 16 + 16 * i + j];
        }
        for (int j = 0; j < LAPLACE_R; j++) {
            kernel[numOctaves * 12 * 16 + 16 * i + j] /= kernelSum;
        }
        scale *= diffScale;
    }
}

void ExtractSift(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, 
        float thresh, float lowestScale, bool scaleUp, float *tempMemory) {
    TimerGPU timer(0);
    unsigned int *d_PointCounterAddr;
    safeCall(hipGetSymbolAddress((void **)&d_PointCounterAddr, d_PointCounter));
    safeCall(hipMemset(d_PointCounterAddr, 0, (8 * 2 + 1) * sizeof(int)));
    safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &siftData.maxPts, sizeof(int)));

    const int nd = NUM_SCALES + 3;
    int w = img.width * (scaleUp ? 2 : 1);
    int h = img.height * (scaleUp ? 2 : 1);
    int p = iAlignUp(w, 128);
    int width = w, height = h;
    int size = h * p;
    int sizeTmp = nd * h * p;
    for (int i = 0; i < numOctaves; i++) {
        w /= 2;
        h /= 2;
        int p = iAlignUp(w, 128);
        size += h * p;
        sizeTmp += nd * h * p;
    }
    float *memoryTmp = tempMemory;
    size += sizeTmp;
    if (!tempMemory) {
        size_t pitch;
        safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size + 4095)/4096 * sizeof(float)));
#ifdef VERBOSE
        printf("Allocated memory size: %d bytes\n", size);
        printf("Memory allocated time = \t %.2f ms \n\n", timer.read());
#endif
    }

    float *memorySub = memoryTmp + sizeTmp;

    CudaImage lowImg;
    lowImg.Allocate(width, height, iAlignUp(width, 128), false, memorySub);
    if (!scaleUp) {
        float kernel[8 * 12 * 16];
        PrepareLaplaceKernels(numOctaves, 0.0f, kernel);
        safeCall(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_LaplaceKernel), kernel, 8 * 12 * 16 * sizeof(float)));
        LowPass(lowImg, img, max(initBlur, 0.001f));
        TimerGPU timer1(0);
        ExtractSiftLoop(siftData, lowImg, numOctaves, 0.0f, thresh, lowestScale, 1.0f, memoryTmp, memorySub + height * iAlignUp(width, 128));
        safeCall(hipMemcpy(&siftData.numPts, &d_PointCounterAddr[2 * numOctaves], sizeof(int), hipMemcpyDeviceToHost));
        siftData.numPts = (siftData.numPts<siftData.maxPts ? siftData.numPts : siftData.maxPts);
        printf("SIFT extraction time = \t %.2f ms %d\n", timer1.read(), siftData.numPts);
    } else {
        CudaImage upImg;

    }

    if (!tempMemory) {
        safeCall(hipFree(memoryTmp));
    }
#ifdef MANAGEDMEN
    safeCall(hipDeviceSynchronize());
#else
    if (siftData.h_data) {
        safeCall(hipMemcpy(siftData.h_data, siftData.d_data, sizeof(SiftPoint) * siftData.numPts, hipMemcpyDeviceToHost));
    }
#endif
    double totTime = timer.read();
    printf("Incl prefiltering & memcpy = \t %.2f ms %d\n\n", totTime, siftData.numPts);
}

void InitSiftData(SiftData &data, int num, bool host, bool dev) {
    data.numPts = 0;
    data.maxPts = num;
    int sz = sizeof(SiftPoint) * num;
#ifdef MANAGEDMEN
    safeCall(hipMallocManaged((void **) &data.m_data, sz));
#else
    data.h_data = nullptr;
    if (host) {
        data.h_data = (SiftPoint *)malloc(sz);
    }
    data.d_data = nullptr;
    if (dev) {
        safeCall(hipMalloc((void **)&data.d_data, sz));
    }
#endif
}

double LowPass(CudaImage &res, CudaImage &src, float scale) {

    float kernel[2 * LOWPASS_R + 1];
    static float oldScale = -1.0f;
    if (scale != oldScale) {
        float kernelSum = 0.0f;
        float ivar2 = 1.0f / (2.0f * scale * scale);
        for (int j = -LOWPASS_R; j <= LOWPASS_R; j++) {
            kernel[j + LOWPASS_R] = (float)expf(-(double)j * j * ivar2);
            kernelSum += kernel[j + LOWPASS_R];
        }
        for (int j = -LOWPASS_R; j <= LOWPASS_R; j++) {
            kernel[j + LOWPASS_R] /= kernelSum;
        }
        safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_LowPassKernel), kernel, (2 * LOWPASS_R + 1) * sizeof(float)));
        oldScale = scale;
    }

    int width = res.width;
    int pitch = res.pitch;
    int height = res.height;
    dim3 blocks(iDivUp(width, LOWPASS_W), iDivUp(height, LOWPASS_H));
#if 1
    dim3 threads(LOWPASS_W + 2 * LOWPASS_R, 4);
    LowPassBlock<<<blocks, threads>>>(src.d_data, res.d_data, width, pitch, height);
#else
    LowPass<<<blocks, threads>>>(src.d_data, res.d_data, width, pitch, height);
#endif
    checkMsg("LowPass() execution failed\n");
    return 0.0;
}


int ExtractSiftLoop(SiftData &siftData, CudaImage &img, int numOctaves, double initBlur, float thresh,
        float lowestScale, float subsampling, float *memoryTmp, float *memorySub) {
#ifdef VERBOSE
    TimerGPU timer(0);
#endif
    int w = img.width;
    int h = img.height;
    if (numOctaves > 1) {
        CudaImage subImg;
        int p = iAlignUp(w / 2, 128);
        subImg.Allocate(w / 2, h / 2, p, false, memorySub);
        ScaleDown(subImg, img, 0.5f);
        float totInitBlur = (float)sqrt(initBlur * initBlur + 0.5f * 0.5f) / 2.0f;
        ExtractSiftLoop(siftData, subImg, numOctaves - 1, totInitBlur, thresh, lowestScale, subsampling * 2.0f, memoryTmp, memorySub + (h / 2) * p);
    }
    ExtractSiftOctave(siftData, img, numOctaves, thresh, lowestScale, subsampling, memoryTmp);
#ifdef VERBOSE
    double totTime = timer.read();
    printf("ExtractSift time total = \t %.2f ms %d\n\n", totTime, numOctaves);
#endif
    return 0;
}

void ExtractSiftOctave(SiftData &siftData, CudaImage &img, int octave, float thresh, 
        float lowestScale, float subsampling, float *memoryTmp) {
    const int nd = NUM_SCALES + 3;
#ifdef VERBOSE
    unsigned int *d_PointCounterAddr;
    safeCall(hipGetSymbolAddress((void**)&d_PointCounterAddr, d_PointCounter));
    unsigned int fstPts, totPts;
    safeCall(hipMemcpy(&fstPts, &d_PointCounterAddr[2 * octave-1], sizeof(int), hipMemcpyDeviceToHost)); 
    TimerGPU timer0;
#endif
    CudaImage diffImg[nd];
    int w = img.width; 
    int h = img.height;
    int p = iAlignUp(w, 128);
    for (int i=0;i<nd-1;i++) {
        diffImg[i].Allocate(w, h, p, false, memoryTmp + i*p*h); 
    }

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = img.d_data;
    resDesc.res.pitch2D.width = img.width;
    resDesc.res.pitch2D.height = img.height;
    resDesc.res.pitch2D.pitchInBytes = img.pitch * sizeof(float);  
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

#ifdef VERBOSE
    TimerGPU timer1;
#endif
    float baseBlur = pow(2.0f, -1.0f / NUM_SCALES);
    float diffScale = pow(2.0f, 1.0f / NUM_SCALES);
    LaplaceMulti(texObj, img, diffImg, octave); 
    FindPointsMulti(diffImg, siftData, thresh, 10.0f, 1.0f / NUM_SCALES, lowestScale / subsampling, subsampling, octave);
#ifdef VERBOSE
    double gpuTimeDoG = timer1.read();
    TimerGPU timer4;
#endif
    ComputeOrientations(texObj, img, siftData, octave); 
    ExtractSiftDescriptors(texObj, siftData, subsampling, octave); 
    //OrientAndExtract(texObj, siftData, subsampling, octave); 
  
    safeCall(hipDestroyTextureObject(texObj));
#ifdef VERBOSE
    double gpuTimeSift = timer4.read();
    double totTime = timer0.read();
    printf("GPU time : %.2f ms + %.2f ms + %.2f ms = %.2f ms\n", totTime-gpuTimeDoG-gpuTimeSift, gpuTimeDoG, gpuTimeSift, totTime);
    safeCall(hipMemcpy(&totPts, &d_PointCounterAddr[2*octave+1], sizeof(int), hipMemcpyDeviceToHost));
    totPts = (totPts<siftData.maxPts ? totPts : siftData.maxPts);
    if (totPts>0) 
        printf("           %.2f ms / DoG,  %.4f ms / Sift,  #Sift = %d\n", gpuTimeDoG/NUM_SCALES, gpuTimeSift/(totPts-fstPts), totPts-fstPts); 
#endif
}

double ScaleDown(CudaImage &res, CudaImage &src, float variance) {
    static float oldVariance = -0.1f;
    if (res.d_data == nullptr || src.d_data == nullptr) {
        printf("ScaleDown: missing data\n");
        return 0.0;
    }
    if (oldVariance != variance) {
        float h_Kernel[5];
        float kernelSum = 0.0f;
        for (int j = 0; j < 5; j++) {
            h_Kernel[j] = (float)expf(-(double)(j - 2) * (j - 2) / 2.0 / variance);
            kernelSum += h_Kernel[j];
        }
        for (int j = 0; j < 5; j++) {
            h_Kernel[j] /= kernelSum;
        }
        safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_ScaleDownKernel), h_Kernel, 5 * sizeof(float)));
        oldVariance = variance;
    }

#if 0
    dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
    dim3 threads(SCALEDOWN_W + 4, SCALEDOWN_H + 4);
    ScaleDownDenseShift<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch);
#else
    dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
    dim3 threads(SCALEDOWN_W + 4);
    ScaleDown<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch);
#endif
    checkMsg("ScaleDown() execution failed\n");
    return 0.0;
}

double LaplaceMulti(hipTextureObject_t texObj, CudaImage &baseImage, CudaImage * results, int octave) {

    int width = results[0].width;
    int pitch = results[0].pitch;
    int height = results[0].height;
#if 1
    dim3 threads(LAPLACE_W + 2 * LAPLACE_R);
    dim3 blocks(iDivUp(width, LAPLACE_W), height);
    LaplaceMultiMem<<<blocks, threads>>>(baseImage.d_data, results[0].d_data, width, pitch, height, octave);
#endif
    checkMsg("LaplaceMulti() execution failed\n");
    return 0.0;
}

double FindPointsMulti(CudaImage *sources, SiftData &siftData, float thresh, float edgeLimit,
        float factor, float lowestScale, float subsampling, int octave) {
    if (sources->d_data == nullptr) {
        printf("FindPointsMulti: missing data\n");
        return 0.0;
    }
    int w = sources->width;
    int p = sources->pitch;
    int h = sources->height;
#if 1
    dim3 blocks(iDivUp(w, MINMAX_W) * NUM_SCALES, iDivUp(h, MINMAX_H));
    dim3 threads(MINMAX_W + 2);
#ifdef MANAGEDMEM
    FindPointsMulti<<<blocks, threads>>>(sources->d_data, siftData.m_data, w, p, h, 
            subsampling, lowestScale, thresh, factor, edgeLimit, octave);
#else
    FindPointsMultiNew<<<blocks, threads>>>(sources->d_data, siftData.d_data, w, p, h,
            subsampling, lowestScale, thresh, factor, edgeLimit, octave);
#endif
#endif
    checkMsg("FindPointsMulti() execution failed\n");
    return 0.0;
}

double ExtractSiftDescriptors(hipTextureObject_t texObj, SiftData &siftData, float subsampling, int octave) {

    dim3 blocks(512);
    dim3 threads(16, 8);
#ifdef MANAGEDMEM
    ExtractSiftDescriptorsCONST<<<blocks, threads>>>(texObj, siftData.m_data, subsampling, octave);
/* #else */
/*     ExtractSiftDescriptorsCONSTNew<<<blocks, threads>>>(texObj, siftData.d_data, subsampling, octave); */
#endif
    checkMsg("ExtractSiftDescriptors() execution failed\n");
    return 0.0;
}


double ComputeOrientations(hipTextureObject_t texObj, CudaImage &src, SiftData &siftData, int octave) {
    dim3 blocks(512); 
#ifdef MANAGEDMEM
    ComputeOrientationsCONST<<<blocks, threads>>>(texObj, siftData.m_data, octave);
#else
#if 1
    dim3 threads(11*11);
    ComputeOrientationsCONST<<<blocks, threads>>>(texObj, siftData.d_data, octave);
#else
    dim3 threads(256); 
    ComputeOrientationsCONSTNew<<<blocks, threads>>>(src.d_data, src.width, src.pitch, src.height, siftData.d_data, octave);
#endif
#endif
    checkMsg("ComputeOrientations() execution failed\n");
    return 0.0;
}

