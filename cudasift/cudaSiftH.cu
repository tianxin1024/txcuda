#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaSift.h"
#include "cudautils.h"


void InitCuda(int devNum) {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if (!nDevices) {
        std::cerr << "No CUDA devices available" << std::endl;
        return ;
    }

    devNum = std::min(nDevices - 1, devNum);
    deviceInit(devNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, devNum);
    std::cout << "\t Device Number: " << devNum << std::endl;
    std::cout << "\t Memory name: " << prop.name << std::endl;
    std::cout << "\t Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
    printf("\t Peak Memory Bandwidth (GB/s): %.1f\n\n",
	            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    return ;
}

