#include <cstdio>
#include "cudautils.h"
#include "cudaImage.h"

void CudaImage::Allocate(int w, 
                         int h, 
                         int p, 
                         bool host, 
                         float *devmem, 
                         float *hostmem) {
    width = w;
    height = h;
    pitch = p;
    d_data = devmem;
    h_data = hostmem;
    t_data = nullptr;

    if (devmem == nullptr) {
        safeCall(hipMallocPitch((void **)&d_data, (size_t*)&pitch, (size_t)(sizeof(float) * width), (size_t)height));
        pitch /= sizeof(float);
        if (d_data == nullptr) {
            printf("Failed to allocate device data\n");
            d_internalAlloc = true;
        }

        if (host && hostmem == nullptr) {
            h_data = (float *)malloc(sizeof(float) * pitch * height);
            h_internalAlloc = true;
        }
    }
}

CudaImage::CudaImage() : width(0), height(0), d_data(nullptr), t_data(nullptr), 
                d_internalAlloc(false), h_internalAlloc(false) {}

CudaImage::~CudaImage() {
    if (d_internalAlloc && d_data != nullptr) {
        safeCall(hipFree(d_data));
    }
    d_data = nullptr;
    if (h_internalAlloc && h_data != nullptr) {
        free(h_data);
    }
    h_data = nullptr;
    if (t_data != nullptr) {
        safeCall(hipFreeArray((hipArray *)t_data));
    }
    t_data = nullptr;
}


double CudaImage::Download() {
    TimerGPU timer(0);
    int p = sizeof(float) * pitch;
    if (d_data != nullptr && h_data != nullptr) {
        safeCall(hipMemcpy2D(d_data, p, h_data, sizeof(float) * width, sizeof(float) * width, height, hipMemcpyHostToDevice));
    }
    double gpuTime = timer.read();
#ifdef VERBOSE
    printf("Download  time = \t \t %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}

double CudaImage::Readback() {
    TimerGPU timer(0);
    int p = sizeof(float) * pitch;
    safeCall(hipMemcpy2D(h_data, sizeof(float) * width, d_data, p, sizeof(float) * width, height, hipMemcpyDeviceToHost));
    double gpuTime = timer.read();
#ifndef VERBOSE
    printf("Readback time = \t\t %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}

double CudaImage::InitTexture() {
    TimerGPU timer(0);
    hipChannelFormatDesc t_desc = hipCreateChannelDesc<float>();
    safeCall(hipMallocArray((hipArray **)&t_data, &t_desc, pitch, height));
    if (t_data == nullptr) {
        printf("Failed to allocated texture data\n");
    }
    double gpuTime = timer.read();
#ifdef VERBOSE
    printf("InitTexture time = \t\t  %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}


double CudaImage::CopyToTexture(CudaImage &dst, bool host) {
    if (dst.t_data == nullptr) {
        printf("Error CopyToTexture: No texture data\n");
        return 0.0;
    }
    if ((!host || h_data == nullptr) && (host || d_data == nullptr)) {
        printf("Error CopyToTexture: No source data\n");
        return 0.0;
    }
    TimerGPU timer(0);
    if (host) {
        safeCall(hipMemcpyToArray((hipArray *)dst.t_data, 0, 0, h_data, 
                    sizeof(float) * pitch * dst.height, hipMemcpyHostToDevice));
    } else {
        safeCall(hipMemcpyToArray((hipArray *)dst.t_data, 0, 0, d_data,
                sizeof(float) * pitch * dst.height, hipMemcpyDeviceToHost));
    }
    safeCall(hipDeviceSynchronize());
    double gpuTime = timer.read();
#ifdef VERBOSE
    printf("CopyToTexture  time = \t\t  %.2f ms\n", gpuTime);
#endif
    return gpuTime;
}


int iDivUp(int a, int b) {
    return (a%b != 0) ? (a/b + 1) : (a/b);
}

int iDivDown(int a, int b) {
    return a / b;
}

int iAlignUp(int a, int b) {
    return (a%b != 0) ? (a - a % b + b) : a;
}

int iAlignDown(int a, int b) {
    return a - a%b;
}

