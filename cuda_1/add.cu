#include "hip/hip_runtime.h"
#include "add.cuh"

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}


int add(int a, int b) {
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    add<<<1, 1>>>(a, b, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    return c;
}
